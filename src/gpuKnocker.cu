/**
 * Starts the actual algorithm.
 */

#include "gpuKnocker.cuh"

#include <stdio.h>
#include <stdlib.h>

#include "cudaCheck.cuh"
#include "evolutionaryAlgorithm.cuh"
#include "lpSolver.cuh"
#include "parameters.cuh"
#include "statistics.cuh"

char *knock(char *mps, char *target, char *parameter) {
	Parameters *parameters = createParameters();

	parseParameters(parameter, mps, target, parameters);
	if (parameters->isVerbose) {
		printParameters(parameters);
	}

	hipEvent_t custart, custop;
	cudaCheck(hipEventCreate(&custart));
	cudaCheck(hipEventCreate(&custop));
	cudaCheck(hipEventRecord(custart, 0));

	EvolutionaryAlgorithm *evolutionaryAlgorithm = createEvolutionaryAlgorithm(
			parameters);
	LPSolver *lpSolver = createLPSolver(parameters);
	Statistics *statistics = createStatistics(parameters);

	preprocessLPProblem(lpSolver, parameters);
	char *knockouts = runEvolutionaryAlgorithm(evolutionaryAlgorithm, lpSolver,
			statistics, parameters);
	if (parameters->isVerbose) {
		printStatistics(statistics, parameters);
	}

	deleteEvolutionaryAlgorithm(evolutionaryAlgorithm);
	deleteLPSolver(lpSolver);
	deleteStatistics(statistics);

	cudaCheck(hipEventRecord(custop, 0));
	cudaCheck(hipEventSynchronize(custop));
	float elapsedTime;
	cudaCheck(hipEventElapsedTime(&elapsedTime, custart, custop));
	if (parameters->isBenchmark) {
		printf("%3.1f\n", elapsedTime);
	} else {
		printf("This took %3.1f ms.\n", elapsedTime);
	}
	cudaCheck(hipEventDestroy(custart));
	cudaCheck(hipEventDestroy(custop));

	deleteParameters(parameters);

	return knockouts;
}
