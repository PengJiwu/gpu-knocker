/**
 * Implementation of evolutionary algorithm.
 */

#include "evolutionaryAlgorithm.cuh"

#include <stdlib.h>
#include <string.h>

#include "cudaCheck.cuh"
#include "eaKernels.cuh"
#include "lpKernels.cuh"

EvolutionaryAlgorithm *createEvolutionaryAlgorithm(Parameters *parameters) {
	EvolutionaryAlgorithm *evolutionaryAlgorithm =
			(EvolutionaryAlgorithm *) malloc(sizeof(EvolutionaryAlgorithm));
	cudaCheck(
			hipMalloc(&evolutionaryAlgorithm->fitness,
					parameters->individualSizeInt * parameters->populationSize
							* sizeof(uint32_t)));
	cudaCheck(
			hipMalloc(&evolutionaryAlgorithm->population,
					parameters->individualSizeInt * parameters->populationSize
							* sizeof(uint32_t)));
	cudaCheck(
			hipMalloc(&evolutionaryAlgorithm->rngState,
					parameters->blockSize * parameters->gridSize
							* sizeof(hiprandStatePhilox4_32_10)));
	cudaCheck(
			hipMalloc(&evolutionaryAlgorithm->temporaryPopulation,
					parameters->individualSizeInt * parameters->populationSize
							* sizeof(uint32_t)));

	return evolutionaryAlgorithm;
}

void deleteEvolutionaryAlgorithm(EvolutionaryAlgorithm *evolutionaryAlgorithm) {
	cudaCheck(hipFree(evolutionaryAlgorithm->fitness));
	cudaCheck(hipFree(evolutionaryAlgorithm->population));
	cudaCheck(hipFree(evolutionaryAlgorithm->rngState));
	cudaCheck(hipFree(evolutionaryAlgorithm->temporaryPopulation));
	free(evolutionaryAlgorithm);
}

void evaluatePopulation(LPSolver *lpSolver,
		EvolutionaryAlgorithm *evolutionaryAlgorithm,
		const Parameters * const parameters) {
	printf("DUMMY evaluatePopulation\n");
	solveLP<<<parameters->gridSize, parameters->blockSize>>>(
			evolutionaryAlgorithm->population, evolutionaryAlgorithm->fitness);
}

char *runEvolutionaryAlgorithm(EvolutionaryAlgorithm *evolutionaryAlgorithm,
		LPSolver *lpSolver, Statistics *statistics,
		const Parameters * const parameters) {
	initializeRNG<<<parameters->gridSize, parameters->blockSize>>>(
			evolutionaryAlgorithm->rngState);
	createPopulation<<<parameters->gridSize, parameters->blockSize>>>(
			evolutionaryAlgorithm->population, evolutionaryAlgorithm->rngState);
	evaluatePopulation(lpSolver, evolutionaryAlgorithm, parameters);

	for (uint32_t iteration = 0; iteration < parameters->iterationAmount;
			iteration++) {
		// migration after specified interval
		if ((iteration + 1) % parameters->migrationInterval == 0) {
			migratePopulation<<<parameters->gridSize, parameters->blockSize>>>(
					evolutionaryAlgorithm->population,
					statistics->iterationData, iteration);
		}

		selectPopulation<<<parameters->gridSize, parameters->blockSize>>>(
				evolutionaryAlgorithm->population,
				evolutionaryAlgorithm->temporaryPopulation,
				evolutionaryAlgorithm->fitness,
				evolutionaryAlgorithm->rngState);
		crossoverPopulation<<<parameters->gridSize, parameters->blockSize>>>(
				evolutionaryAlgorithm->population,
				evolutionaryAlgorithm->temporaryPopulation,
				evolutionaryAlgorithm->rngState);
		hipDeviceSynchronize();
		swapTemporaryPopulation(&evolutionaryAlgorithm->population,
				&evolutionaryAlgorithm->temporaryPopulation);

		mutatePopulation<<<parameters->gridSize, parameters->blockSize>>>(
				evolutionaryAlgorithm->population,
				evolutionaryAlgorithm->rngState);

		hipDeviceSynchronize();

		evaluatePopulation(lpSolver, evolutionaryAlgorithm, parameters);
		gatherStatistics(statistics, evolutionaryAlgorithm->fitness, iteration,
				parameters);
	}

	char *knockouts = (char *) malloc(18 * sizeof(char));
	strcpy(knockouts, "42,DUMMY,knockout");
	return knockouts;
}

void swapTemporaryPopulation(uint32_t **population,
		uint32_t **temporaryPopulation) {
	uint32_t *temp = *population;
	*population = *temporaryPopulation;
	*temporaryPopulation = temp;
}
